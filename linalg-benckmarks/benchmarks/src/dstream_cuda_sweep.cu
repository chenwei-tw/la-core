#include "hip/hip_runtime.h"
//==========================================================================
// Copyright (c) 2017 Samuel Steffl
// All rights reserved.
//
// CUDA implementation of the HPCC STREAM benchmark
// http://www.cs.virginia.edu/~mccalpin/papers/bandwidth/node2.html
// http://www.cs.virginia.edu/stream/ref.html
//
// For bandwidth, using the 2st of the 3 methods, counting reads and writes 
// as separate transfers.
//=========================================================================

#include <assert.h>
#include <stdbool.h>
#include <string.h>
#include <time.h>

extern "C" {
#include "get_real_time.h"
#include "matrix_types.h"
#include "matrix_utilities.h"
#include "util.h"
}

//==========================================================================

static const IDX VEC_MIN       = (1 << 16);
static const IDX VEC_MAX       = (1 << 20);
static const IDX VEC_SCALE     = 2;

static const double q          = 2.5;

//==========================================================================
// COPY
//==========================================================================

void do_copy(IDX size)
{
  //--------------------------- CONFIG -------------------------------
  double *A = create_matrix_dr(size, 1, NO_FILL);
  double *B = create_matrix_dr(size, 1, FILL);

  //---------------------------- RUN ------------------------------------
  double start = get_real_time();
  for(IDX i=0; i<size; i+=1) {
    A[i] = B[i];
  }
  double duration = get_real_time() - start;

  //--------------------------- REPORT -----------------------------------
  double flop = STREAM_COPY_FLOP(size);
  double bytes = STREAM_COPY_DP_BYTES(size);
  printf("COPY  SIZE=%10d | took % 10.6f secs | ", size, duration);
  printf("% 10.3f MB/s | ", bytes/(duration*1e6));
  printf("% 10.3f MFLOP/s\n", flop/(duration*1e6));

  //--------------------------- CLEANUP ----------------------------------
  free_matrix_d(A);
  free_matrix_d(B);
}

//==========================================================================
// SCALE
//==========================================================================

__global__ void DoScaleKernel(double *dA, double *dB, IDX count)
{
  int start = blockIdx.x*blockDim.x + threadIdx.x;
  int end = start + count;

  for (IDX i=start; i<end; ++i) {
    dA[i] = q*dB[i];
  }
}

//minimum of 4096 elements
void do_cuda_scale(double *A, double *B, IDX size)
{
  const IDX BLOCK_SIZE = 256;
  const IDX IDX_PER_THREAD = MIN(size/BLOCK_SIZE, 16);

  double *dA, *dB;
  IDX A_size = size*sizeof(double);
  IDX B_size = size*sizeof(double);

  //allocate matrices on device
  hipMalloc((void **)&dA, A_size);
  hipMalloc((void **)&dB, B_size);

  //transfer A, to device
  hipMemcpy(dB, B, B_size, hipMemcpyHostToDevice);

  //perform kernel
  assert(size % BLOCK_SIZE*IDX_PER_THREAD == 0);
  dim3 dimGrid(size/BLOCK_SIZE/IDX_PER_THREAD, 1);
  dim3 dimBlock(BLOCK_SIZE, 1);
  DoScaleKernel<<<dimGrid, dimBlock>>>(dA, dB, IDX_PER_THREAD);

  //transfer A to host
  hipMemcpy(A, dA, A_size, hipMemcpyDeviceToHost);  

  //free matrices on device
  hipFree(dA);
  hipFree(dB);
}

//According to book, limits are:
//  up to 512 threads per block
//  up to 8 blocks per SM at a time
//  up to 1024 threads per SM
//so: blocks should be 256x1 threads (256 threads) which satisfies all 3 above
void do_scale(IDX size)
{
  //--------------------------- CONFIG -------------------------------
  double *A = create_matrix_dr(size, 1, NO_FILL);
  double *B = create_matrix_dr(size, 1, FILL);

  //---------------------------- RUN ------------------------------------
  double start = get_real_time();
  do_cuda_scale(A, B, size);
  double duration = get_real_time() - start;

  //--------------------------- REPORT -----------------------------------
  double flop = STREAM_SCALE_FLOP(size);
  double bytes = STREAM_SCALE_DP_BYTES(size);
  printf("SCALE SIZE=%10d | took % 10.6f secs | ", size, duration);
  printf("% 10.3f MB/s | ", bytes/(duration*1e6));
  printf("% 10.3f MFLOP/s\n", flop/(duration*1e6));

  //--------------------------- CLEANUP ----------------------------------
  free_matrix_d(A);
  free_matrix_d(B);
}

//==========================================================================
// SUM
//==========================================================================

__global__ void DoSumKernel(double *dA, double *dB, double *dC, IDX count)
{
  int start = blockIdx.x*blockDim.x + threadIdx.x;
  int end = start + count;

  for (IDX i=start; i<end; ++i) {
    dA[i] = dB[i] + dC[i];
  }
}

//minimum of 4096 elements
void do_cuda_sum(double *A, double *B, double *C, IDX size)
{
  const IDX BLOCK_SIZE = 256;
  const IDX IDX_PER_THREAD = MIN(size/BLOCK_SIZE, 16);

  double *dA, *dB, *dC;
  IDX A_size = size*sizeof(double);
  IDX B_size = size*sizeof(double);
  IDX C_size = size*sizeof(double);

  //allocate matrices on device
  hipMalloc((void **)&dA, A_size);
  hipMalloc((void **)&dB, B_size);
  hipMalloc((void **)&dC, C_size);

  //transfer B, C, to device
  hipMemcpy(dB, B, B_size, hipMemcpyHostToDevice);
  hipMemcpy(dC, C, C_size, hipMemcpyHostToDevice);

  //perform kernel
  assert(size % BLOCK_SIZE*IDX_PER_THREAD == 0);
  dim3 dimGrid(size/BLOCK_SIZE/IDX_PER_THREAD, 1);
  dim3 dimBlock(BLOCK_SIZE, 1);
  DoSumKernel<<<dimGrid, dimBlock>>>(dA, dB, dC, IDX_PER_THREAD);

  //transfer A to host
  hipMemcpy(A, dA, A_size, hipMemcpyDeviceToHost);   

  //free matrices on device
  hipFree(dA);
  hipFree(dB);
  hipFree(dC);
}

//According to book, limits are:
//  up to 512 threads per block
//  up to 8 blocks per SM at a time
//  up to 1024 threads per SM
//so: blocks should be 256x1 threads (256 threads) which satisfies all 3 above
void do_sum(IDX size)
{
  //--------------------------- CONFIG -------------------------------
  double *A         = create_matrix_dr(size, 1, NO_FILL);
  double *B         = create_matrix_dr(size, 1, FILL);
  double *C         = create_matrix_dr(size, 1, FILL);

  //---------------------------- RUN ------------------------------------
  double start = get_real_time();
  do_cuda_sum(A, B, C, size);
  double duration = get_real_time() - start;

  //--------------------------- REPORT -----------------------------------
  double flop = STREAM_SUM_FLOP(size);
  double bytes = STREAM_SUM_DP_BYTES(size);
  printf("SUM   SIZE=%10d | took % 10.6f secs | ", size, duration);
  printf("% 10.3f MB/s | ", bytes/(duration*1e6));
  printf("% 10.3f MFLOP/s\n", flop/(duration*1e6));

  //--------------------------- CLEANUP ----------------------------------
  free_matrix_d(A);
  free_matrix_d(B);
  free_matrix_d(C);
}

//==========================================================================
// TRIAD
//==========================================================================

__global__ void DoTriadKernel(double *dA, double *dB, double *dC, IDX count)
{
  int start = blockIdx.x*blockDim.x + threadIdx.x;
  int end = start + count;

  for (IDX i=start; i<end; ++i) {
    dA[i] = dB[i] + q*dC[i];
  }
}

//minimum of 4096 elements
void do_cuda_triad(double *A, double *B, double *C, IDX size)
{
  const IDX BLOCK_SIZE = 256;
  const IDX IDX_PER_THREAD = MIN(size/BLOCK_SIZE, 16);

  double *dA, *dB, *dC;
  IDX A_size = size*sizeof(double);
  IDX B_size = size*sizeof(double);
  IDX C_size = size*sizeof(double);

  //allocate matrices on device
  hipMalloc((void **)&dA, A_size);
  hipMalloc((void **)&dB, B_size);
  hipMalloc((void **)&dC, C_size);

  //transfer B, C, to device
  hipMemcpy(dB, B, B_size, hipMemcpyHostToDevice);
  hipMemcpy(dC, C, C_size, hipMemcpyHostToDevice);

  //perform kernel
  assert(size % BLOCK_SIZE*IDX_PER_THREAD == 0);
  dim3 dimGrid(size/BLOCK_SIZE/IDX_PER_THREAD, 1);
  dim3 dimBlock(BLOCK_SIZE, 1);
  DoTriadKernel<<<dimGrid, dimBlock>>>(dA, dB, dC, IDX_PER_THREAD);

  //transfer A to host
  hipMemcpy(A, dA, A_size, hipMemcpyDeviceToHost);   

  //free matrices on device
  hipFree(dA);
  hipFree(dB);
  hipFree(dC);
}

void do_triad(IDX size)
{
  //--------------------------- CONFIG -------------------------------
  double *A         = create_matrix_dr(size, 1, NO_FILL);
  double *B         = create_matrix_dr(size, 1, FILL);
  double *C         = create_matrix_dr(size, 1, FILL);

  //---------------------------- RUN ------------------------------------
  double start = get_real_time();
  do_cuda_triad(A, B, C, size);
  double duration = get_real_time() - start;

  //--------------------------- REPORT -----------------------------------
  double flop = STREAM_TRIAD_FLOP(size);
  double bytes = STREAM_TRIAD_DP_BYTES(size);
  printf("TRIAD SIZE=%10d | took % 10.6f secs | ", size, duration);
  printf("% 10.3f MB/s | ", bytes/(duration*1e6));
  printf("% 10.3f MFLOP/s\n", flop/(duration*1e6));

  //--------------------------- CLEANUP ----------------------------------
  free_matrix_d(A);
  free_matrix_d(B);
  free_matrix_d(C);
}

//==========================================================================
// MAIN
//==========================================================================

int main(int argc, char **argv)
{
  //---------------------------------------------------------------------------
  unsigned seed = time(NULL);
  IDX size_start = VEC_MIN;
  IDX size_end = VEC_MAX;

  unsigned tmp_seed;
  IDX tmp_log_size;

  //---------------------------------------------------------------------------
  //parse args
  for(int i=1; i<argc; ++i){
    if(sscanf(argv[i], "--seed=%u", &tmp_seed) == 1){
      seed = tmp_seed;
    }
    else if(sscanf(argv[i], "--log_size=%u", &tmp_log_size) == 1){
      size_start = (1 << tmp_log_size);
      size_end = (1 << tmp_log_size);
    }
    else {
      printf("unrecognized argument: %s\n", argv[i]);
      exit(0);
    }
  }

  //---------------------------------------------------------------------------
  //just to kinda randomize the matrices up
  srand(seed);

  //---------------------------------------------------------------------------

  //set breakpoint here, then make command window huge
  for(IDX SIZE=size_start; SIZE <= size_end; SIZE *= VEC_SCALE){
    do_copy(SIZE);
    do_scale(SIZE);
    do_sum(SIZE);
    do_triad(SIZE);
    PRINT("\n");
  }

  //set breakpoint here, then analyze results
  return 0;
}

