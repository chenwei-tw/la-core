#include "hip/hip_runtime.h"
//==========================================================================
// Copyright (c) 2017 Samuel Steffl
// All rights reserved.
//=========================================================================

#include <assert.h>
#include <stdbool.h>
#include <string.h>
#include <time.h>

#include <gsl/gsl_matrix.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

extern "C" {
#include "get_real_time.h"
#include "matrix_types.h"
#include "matrix_utilities.h"
#include "util.h"
#include "verify/common.h"
}

//==========================================================================

static const IDX NMP_MIN        = 128; //was 2
static const IDX NMP_MAX        = 1024;
static const IDX NMP_SCALE      = 2;

static IDX BLOCK_SIZE = 8;

//A trans, B trans
__global__ void Ptrans_kernel(double *dA, double *dB, IDX N)
{
  int i = (blockIdx.x*blockDim.x + threadIdx.x)*BLOCK_SIZE;
  int j = (blockIdx.y*blockDim.y + threadIdx.y)*BLOCK_SIZE;

  //let half the threads just die
  if(j > i) {
    for(IDX ik=0; ik<BLOCK_SIZE; ++ik) {
      for(IDX jk=0; jk<BLOCK_SIZE; ++jk) {
        double tmp = IDXR(dA, j+jk, i+ik, N, N) + IDXR(dB, i+ik, j+jk, N, N);
        IDXR(dA, j+jk, i+ik, N, N) = 
          IDXR(dA, i+ik, j+jk, N, N) + IDXR(dB, j+jk, i+ik, N, N);
        IDXR(dA, i+ik, j+jk, N, N) = tmp;
      }
    }
  }
}

//According to book, limits are:
//  up to 512 threads per block
//  up to 8 blocks per SM at a time
//  up to 1024 threads per SM
//so: blocks should be 16x16 threads (256 threads) (matrices MUST be 16x16 mul)
//    which satisfies all 3 above
static void do_test_cuda(double *A, double *B, IDX N)
{
  IDX BLOCK_SQAR = BLOCK_SIZE*BLOCK_SIZE;
  IDX N_SQAR = N*N;
  IDX MAX_THREADS_PER_BLOCK = 16;

  IDX THREADS_PER_BLOCK = MIN(MAX_THREADS_PER_BLOCK, N/BLOCK_SIZE);
  IDX THREAD_BLOCKS = (N/BLOCK_SIZE)*(N/(BLOCK_SIZE*THREADS_PER_BLOCK));

  double *dC, *dA, *dB;
  IDX C_size = N*M*sizeof(double);
  IDX A_size = N*P*sizeof(double);
  IDX B_size = P*M*sizeof(double);

  //allocate matrices on device
  hipMalloc((void **)&dA, A_size);
  hipMalloc((void **)&dB, B_size);

  //transfer A, B, to device
  hipMemcpy(dA, A, A_size, hipMemcpyHostToDevice);
  hipMemcpy(dB, B, B_size, hipMemcpyHostToDevice);

  //perform kernel
  assert(N % BLOCK == 0);
  dim3 dimGrid(THREAD_BLOCKS, 1);
  dim3 dimBlock(THREADS_PER_BLOCK, 1);
  Ptrans_kernel<<<dimGrid, dimBlock>>>(dA, dB, N);

  //transfer C to host
  hipMemcpy(A, dA, A_size, hipMemcpyDeviceToHost);  

  //free matrices on device
  hipFree(dA);
  hipFree(dB);
}

static void do_test(IDX N)
{
  double *A, *A_cuda, *B;
  B = create_matrix_dr(N, N, FILL);
  A = create_matrix_dr(N, N, FILL);
  A_cuda = copy_matrix_dr(A, N, N);

  if(DEBUG){
    dump_matrix_dr("CUDA: A before", A_cuda, N, N);
    dump_matrix_dr("GSL: A before", A, N, N);
    dump_matrix_dr("GSL: B before", B, N, N);
  }

//==========================================================================
  double start = get_real_time();
  do_test_cuda(A_cuda, B, N);
  double duration = (get_real_time() - start)*1e6;

  double flop = PTRANS_FLOP(N);
  PRINT("N=%4d | CUDA took % 10.6f secs | ",  N, duration);
  PRINT("% 9.3f MFLOP | ", flop / 1e6);
  PRINT("% 12.3f MFLOP/s\n", flop/ duration);

//==========================================================================
  start = get_real_time();
  gsl_matrix_view gsl_A = gsl_matrix_view_array(A, N, N);
  gsl_matrix_view gsl_B = gsl_matrix_view_array(B, N, N);
  gsl_matrix_transpose (&gsl_A.matrix);
  gsl_matrix_add (&gsl_A.matrix, &gsl_B.matrix);
  duration = (get_real_time() - start)*1e6;

  PRINT("N=%4d | GSL took % 10.6f secs | ",  N, duration);
  PRINT("% 9.3f MFLOP | ", flop / 1e6);
  PRINT("% 12.3f MFLOP/s\n", flop/ duration);

  for(IDX i=0; i<N; ++i){
    for(IDX j=0; j<N; ++j){
      double num1 = IDXR(A, i, j, N, N);
      double num2 = IDXR(A_cuda, i, j, N, N);
      if(MARGIN_EXCEEDED(num1, num2)){
        printf("Matrices differ at [%d,%d]\n", i, j);
        dump_matrix_dr("CUDA: A after", A_cuda, N, N);
        dump_matrix_dr("GSL: A after", A, N, N);
        return;
      }
    }
  }
  printf("matrix correct\n");

  if(DEBUG){
    dump_matrix_dr("CUDA: A after", A_cuda, N, N);
    dump_matrix_dr("GSL: A after", A, N, N);
  }

  free_matrix_d(A_cuda);
  free_matrix_d(A);
  free_matrix_d(B);
}

int main(int argc, char **argv)
{
  //---------------------------------------------------------------------------
  unsigned seed = time(NULL);
  IDX nmp_start = NMP_MIN;
  IDX nmp_end = NMP_MAX;

  unsigned tmp_seed;
  IDX tmp_nmp;

  //---------------------------------------------------------------------------
  //parse args
  for(int i=1; i<argc; ++i){
    if(sscanf(argv[i], "--seed=%u", &tmp_seed) == 1){
      seed = tmp_seed;
    }
    else if(!strcmp(argv[i], "--debug")){
      DEBUG = true;
    }
    else if(sscanf(argv[i], "--log_size=%u", &tmp_nmp) == 1){
      nmp_start = (1 << tmp_nmp);
      nmp_end = (1 << tmp_nmp);
    }
    else {
      printf("unrecognized argument: %s\n", argv[i]);
      exit(0);
    }
  }

  //---------------------------------------------------------------------------
  //just to kinda randomize the matrices up
  srand(seed);

  //---------------------------------------------------------------------------

  //set breakpoint here, then make command window huge

  for(IDX NMP=nmp_start; NMP <= nmp_end; NMP *= NMP_SCALE){
    do_test(NMP);
    PRINT("\n");
  }

  //set breakpoint here, then analyze results
  return 0;
}
