#include "hip/hip_runtime.h"
//==========================================================================
// Copyright (c) 2017 Samuel Steffl
// All rights reserved.
//=========================================================================

#include <stdbool.h>
#include <string.h>
#include <time.h>

extern "C" {
#include "get_real_time.h"
#include "matrix_types.h"
#include "matrix_utilities.h"
#include "util.h"
}

static const IDX NP_MIN = 4;
static const IDX NP_MAX = 1024;
static const IDX NP_SCALE = 2;

//==========================================================================

//row major order
static void swap_rows_dr(double *A, IDX h, IDX w, IDX r1, IDX r2)
{
  double *tmp = (double *)malloc(sizeof(double)*w);
  double *r1_start = &(IDXR(A, r1, 0, h, w));
  double *r2_start = &(IDXR(A, r2, 0, h, w));
  memcpy(tmp, r1_start, sizeof(double)*w);
  memcpy(r1_start, r2_start, sizeof(double)*w);
  memcpy(r2_start, tmp, sizeof(double)*w);
}

//col major order
static void swap_rows_dc(double *A, IDX h, IDX w, IDX r1, IDX r2)
{
  for (IDX i = 0; i < w; i++) {
    double tmp = IDXC(A, r1, i, h, w);
    IDXC(A, r1, i, h, w) = IDXC(A, r2, i, h, w);
    IDXC(A, r2, i, h, w) = tmp;
  }
}

//A in row-major, B in col-major order
void permute_in_place(double *A, double *B, IDX iter, IDX N, IDX P)
{
  IDX col = iter;
  double max_val = IDXR(A, col, col, N, N);
  IDX max_val_row = col;
  for(IDX row=col+1; row<N; ++row){
    double new_val = IDXR(A, row, col, N, N);
    if(new_val > max_val){
      max_val = new_val;
      max_val_row = row;
    }
  }
  if(max_val_row != col){
    swap_rows_dr(A, N, N, col, max_val_row);
    swap_rows_dc(B, N, P, col, max_val_row);
  }
}

//==========================================================================
// In place LU Factorization
//==========================================================================

__global__ void l_iteration_in_place(double *A, IDX N, IDX iter)
{
  for (IDX j = (i + 1); j<N; ++j) {
    double *Lji = &(IDXR(A, j, i, N, N));
    double Uii = IDXR(A, i, i, N, N);

    for (IDX k; k<i; ++k) {
      double Ljk = IDXR(A, j, k, N, N);
      double Uki = IDXR(A, k, i, N, N);
      *Lji -= Ljk * Uki;
    }
    *Lji /= Uii;
  }
}

__global__ void u_iteration_in_place(double *A, IDX N, IDX iter)
{
  IDX i = iter;
  IDX j = (blockIdx.x*blockDim.x + threadIdx.x) + iter;

  if(iter > 0) {
    double *Uip1j = &(IDXR(A, i + 1, j, N, N));
    for (IDX k; k<i; ++k) {
      double Lip1k =IDXR(A, i + 1, k, N, N);
      double Ukj = IDXR(A, k, j, N, N);
      *Uip1j -= Lip1k * Ukj;
    }
  }
}

static void do_cuda_lu_solve(double *A, double *B, IDX N, IDX P)
{
  const IDX MAX_THREADS = 512;

  double *dC, *dA, *dB;
  IDX C_size = N*M*sizeof(double);
  IDX A_size = N*P*sizeof(double);
  IDX B_size = P*M*sizeof(double);

  //allocate matrices on device
  hipMalloc((void **)&dC, C_size);
  hipMalloc((void **)&dA, A_size);
  hipMalloc((void **)&dB, B_size);

  //transfer A, B, to device
  hipMemcpy(dA, A, A_size, hipMemcpyHostToDevice);
  hipMemcpy(dB, B, B_size, hipMemcpyHostToDevice);
  hipMemcpy(dC, C, C_size, hipMemcpyHostToDevice);

  //perform kernel
  for (IDX iter = 0; iter < N; ++iter) {
    permute_in_place(A, B, iter, N, P);

    IDX num_threads = MIN(size/2, MAX_THREADS);
    dim3 dimGrid(M/BLOCK_SIZE, 1);
    dim3 dimBlock(BLOCK_SIZE, 1);
    u_iteration_in_place<<<dimGrid,dimBlock>>>(A, N, iter);

    l_iteration_in_place<<<dimGrid,dimBlock>>>(A, N, iter);
  }
  //use_cublas(dC, dA, dB, N, M, P, A_trans, B_trans);

  //transfer C to host
  hipMemcpy(C, dC, C_size, hipMemcpyDeviceToHost);  

  //free matrices on device
  hipFree(dC);
  hipFree(dA);
  hipFree(dB);
}

static void do_test(IDX N, IDX P)
{
  double *A, *B;
  A = create_matrix_dr(N, N, FILL);
  B = create_matrix_dc(N, P, FILL);

  double start = get_real_time();
  do_cuda_lu_solve(A, B, N, P);
  double duration = get_real_time() - start;

  double flop = LU_SOLVE_FLOP(N);
  PRINT("N=%4d P=%4d | took % 10.6f secs | ", N, P, duration);
  PRINT("% 9.3f MFLOP | ", flop / 1e6);
  PRINT("% 12.3f MFLOP/s\n", flop/ duration / 1e6);

  free_matrix_d(A);
  free_matrix_d(B);
}

int main(int argc, char **argv)
{
  //-------------------------------------------------------------------------
  unsigned seed = time(NULL);
  IDX np_start  = NP_MIN;
  IDX np_end    = NP_MAX;

  unsigned tmp_seed;
  IDX tmp_log_size;

  //-------------------------------------------------------------------------
  //parse args
  for(int i=1; i<argc; ++i){
    if(sscanf(argv[i], "--seed=%u", &tmp_seed) == 1){
      seed = tmp_seed;
    }
    else if(sscanf(argv[i], "--log_size=%u", &tmp_log_size) == 1){
      np_start = (1 << tmp_log_size);
      np_end = (1 << tmp_log_size);
    }
    else {
      printf("unrecognized argument: %s\n", argv[i]);
      exit(0);
    }
  }

  //-------------------------------------------------------------------------
  //just to kinda randomize the matrices up
  srand(seed);

  //-------------------------------------------------------------------------

  //set breakpoint here, then make command window huge
  for(IDX NP=np_start; NP <= np_end; NP *= NP_SCALE){
    do_test(NP, NP);
  }

  //set breakpoint here, then analyze results
  return 0;
}
