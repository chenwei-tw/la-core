#include "hip/hip_runtime.h"
//==========================================================================
// Copyright (c) 2017 Samuel Steffl
// All rights reserved.
//
// x86 implementation of the HPCC FFT benchmark
// Naive Algorithm pretty much straight from CLRS Algorithms textbook
//=========================================================================

#include <assert.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdbool.h>
#include <string.h>
#include <time.h>

extern "C" {
#include "get_real_time.h"
#include "matrix_types.h"
#include "matrix_utilities.h"
#include "util.h"
}

//==========================================================================
// Definitions of global control flags
bool DEBUG = false;

#define VEC_MIN 8
#define VEC_MAX 10
#define VEC_SCALE 2


//==========================================================================

// Block index
#define bx blockIdx.x
#define by blockIdx.y
// Thread index
#define tx threadIdx.x

// Possible values are 2, 4, 8 and 16
//#define R 16
#define R 16

inline double2 operator*(double2 a, double2 b)
{
    return make_double2(a.x * b.x - a.y * b.y, a.x * b.y + a.y * b.x);
}

inline double2 operator+(double2 a, double2 b)
{
    return make_double2(a.x + b.x, a.y + b.y);
}

inline double2 operator-(double2 a, double2 b)
{
    return make_double2(a.x - b.x, a.y - b.y);
}

inline double2 operator*(double2 a, double b)
{
    return make_double2(b * a.x , b * a.y);
}

#define COS_PI_8  0.923879533
#define SIN_PI_8  0.382683432
#define exp_1_16  make_double2( COS_PI_8, -SIN_PI_8)
#define exp_3_16  make_double2( SIN_PI_8, -COS_PI_8)
#define exp_5_16  make_double2(-SIN_PI_8, -COS_PI_8)
#define exp_7_16  make_double2(-COS_PI_8, -SIN_PI_8)
#define exp_9_16  make_double2(-COS_PI_8,  SIN_PI_8)
#define exp_1_8   make_double2( 1, -1)
#define exp_1_4   make_double2( 0, -1)
#define exp_3_8   make_double2(-1, -1)

__device__ void GPU_FFT2(double2 &v1, double2 &v2)
{
    double2 v0 = v1;
    v1 = v0 + v2;
    v2 = v0 - v2;
}

__device__ void GPU_FFT4(double2 &v0, double2 &v1, double2 &v2, double2 &v3)
{
    GPU_FFT2(v0, v2);
    GPU_FFT2(v1, v3);
    v3 = v3 * exp_1_4;
    GPU_FFT2(v0, v1);
    GPU_FFT2(v2, v3);
}

inline __device__ void GPU_FFT2(double2* v)
{
    GPU_FFT2(v[0], v[1]);
}

inline __device__ void GPU_FFT4(double2* v)
{
    GPU_FFT4(v[0], v[1], v[2], v[3]);
}

inline __device__ void GPU_FFT8(double2* v)
{
    GPU_FFT2(v[0], v[4]);
    GPU_FFT2(v[1], v[5]);
    GPU_FFT2(v[2], v[6]);
    GPU_FFT2(v[3], v[7]);

    v[5] = (v[5] * exp_1_8) * M_SQRT1_2;
    v[6] = v[6] * exp_1_4;
    v[7] = (v[7] * exp_3_8) * M_SQRT1_2;

    GPU_FFT4(v[0], v[1], v[2], v[3]);
    GPU_FFT4(v[4], v[5], v[6], v[7]);

}

inline __device__ void GPU_FFT16(double2 *v)
{
    GPU_FFT4(v[0], v[4], v[8], v[12]);
    GPU_FFT4(v[1], v[5], v[9], v[13]);
    GPU_FFT4(v[2], v[6], v[10], v[14]);
    GPU_FFT4(v[3], v[7], v[11], v[15]);

    v[5]  = (v[5]  * exp_1_8) * M_SQRT1_2;
    v[6]  =  v[6]  * exp_1_4;
    v[7]  = (v[7]  * exp_3_8) * M_SQRT1_2;
    v[9]  =  v[9]  * exp_1_16;
    v[10] = (v[10] * exp_1_8) * M_SQRT1_2;
    v[11] =  v[11] * exp_3_16;
    v[13] =  v[13] * exp_3_16;
    v[14] = (v[14] * exp_3_8) * M_SQRT1_2;
    v[15] =  v[15] * exp_9_16;

    GPU_FFT4(v[0],  v[1],  v[2],  v[3]);
    GPU_FFT4(v[4],  v[5],  v[6],  v[7]);
    GPU_FFT4(v[8],  v[9],  v[10], v[11]);
    GPU_FFT4(v[12], v[13], v[14], v[15]);
}

__device__ int GPU_expand(int idxL, int N1, int N2)
{
    return (idxL / N1) * N1 * N2 + (idxL % N1);
}

__device__ void GPU_FftIteration(int j, int Ns, double2* data0, double2* data1, int N)
{
    double2 v[R];
    int idxS = j;
    double angle = -2.0 * M_PI * (j % Ns) / (Ns * R);

    for (int r = 0; r < R; r++) {
        v[r] = data0[idxS + r * N / R];
        v[r] = v[r] * make_double2(cos(r * angle), sin(r * angle));
    }

#if R == 2
    GPU_FFT2(v);
#endif

#if R == 4
    GPU_FFT4(v);
#endif

#if R == 8
    GPU_FFT8(v);
#endif

#if R == 16
    GPU_FFT16(v);
#endif

    int idxD = GPU_expand(j, Ns, R);

    for (int r = 0; r < R; r++) {
        data1[idxD + r * Ns] = v[r];
    }

}

__global__ void GPU_FFT_Global(int Ns, double2* data0, double2* data1, int N)
{
    data0 += bx * N;
    data1 += bx * N;
    GPU_FftIteration(tx, Ns, data0, data1, N);
}



//array must be > 512
void do_cuda_fft(double2 *source, double2 *result, IDX size)
{
  IDX MAX_THREADS_PER_BLOCK = 8;
  IDX MIN_THREADS_PER_BLOCK = 8;
  IDX B = MAX(size/R/MAX_THREADS_PER_BLOCK, 1);
  IDX N = size/B;
  if(N/R < MIN_THREADS_PER_BLOCK) {
    N = MIN_THREADS_PER_BLOCK*R;
    B = size/N;
  }
  printf("size,N,R,B = {%d,%d,%d,%d}\n", size, N, R, B);
  assert(B*N == size);

  IDX n_bytes = N * B * sizeof(double2);

   // allocate device memory
  double2 *d_source, *d_work;
  hipMalloc((void**) &d_source, n_bytes);

  // copy host memory to device
  hipMemcpy(d_source, source, n_bytes, hipMemcpyHostToDevice);
  hipMalloc((void**) &d_work, n_bytes);
  hipMemset(d_work, 0, n_bytes);

  for (int Ns = 1; Ns < N; Ns *= R) {
    GPU_FFT_Global <<<dim3(B), dim3(N / R)>>> (Ns, d_source, d_work, N);
    double2 *tmp = d_source;
    d_source = d_work;
    d_work = tmp;

    if(DEBUG){
      printf("d_source=[\n  ");
      for(IDX i=0; i<size;++i){
        printf("{%6.3f,%6.3f},", source[i].x, source[i].y);
        if(i%4 == 3){
          printf("\n  ");
        }
      }
      printf("]\n");
    }
  }
  hipDeviceSynchronize();

  // copy device memory to host
  hipMemcpy(result, d_source, n_bytes, hipMemcpyDeviceToHost);

  //hipFree(d_source);
  //hipFree(d_work);
}

unsigned int my_log2( unsigned int x )
{
  unsigned int ans = 0 ;
  while( x>>=1 ) ans++;
  return ans ;
}

#define FFT_FLOP_GOOD(size) ((double)(5*size*(my_log2(size))))

void do_fft(IDX size)
{
  //--------------------------- CONFIG -------------------------------
  double2 *source = (double2 *)create_matrix_dr(size*2, 1, FILL);
  double2 *result = (double2 *)create_matrix_dr(size*2, 1, NO_FILL);

  for(IDX i=0; i<size; ++i){
    source[i].y = 0.0;
  }

  if(DEBUG){
    printf("source=[\n  ");
    for(IDX i=0; i<size;++i){
      printf("{%6.3f,%6.3f},", source[i].x, source[i].y);
      if(i%4 == 3){
        printf("\n  ");
      }
    }
    printf("]\n");
  }

  //---------------------------- RUN ------------------------------------
  double start = get_real_time();
  do_cuda_fft(source, result, size);
  double duration = get_real_time() - start;

  //--------------------------- REPORT -----------------------------------
  double flop = FFT_FLOP_GOOD(size);
  printf("SIZE=%8d | took % 10.6f secs | ", size, duration);
  printf("% 10.3f MFLOP/s\n", flop/(duration*1e6));

  if(DEBUG){
    printf("result=[\n  ");
    for(IDX i=0; i<size;++i){
      printf("{%6.3f,%6.3f},", result[i].x, result[i].y);
      if(i%4 == 3){
        printf("\n  ");
      }
    }
    printf("]\n");
  }

  //--------------------------- CLEANUP ----------------------------------
  //free_matrix_d((double *)source);
  //free_matrix_d((double *)result);
}



int main(int argc, char **argv)
{
  //---------------------------------------------------------------------------
  unsigned seed = time(NULL);
  IDX size_start = VEC_MIN;
  IDX size_end = VEC_MAX;

  unsigned tmp_seed;
  IDX tmp_log_size;

  //---------------------------------------------------------------------------
  //parse args
  for(int i=1; i<argc; ++i){
    if(sscanf(argv[i], "--seed=%u", &tmp_seed) == 1){
      seed = tmp_seed;
    }
    else if(!strcmp(argv[i], "--debug")){
      DEBUG = true;
    }
    else if(sscanf(argv[i], "--log_size=%u", &tmp_log_size) == 1){
      size_start = 1 << tmp_log_size;
      size_end = 1 << tmp_log_size;
    }
    else {
      printf("unrecognized argument: %s\n", argv[i]);
      exit(0);
    }
  }

  //---------------------------------------------------------------------------
  //just to kinda randomize the matrices up
  srand(seed);

  //---------------------------------------------------------------------------

  //set breakpoint here, then make command window huge
  for(IDX SIZE=size_start; SIZE <= size_end; SIZE *= VEC_SCALE){
    do_fft(SIZE);
    PRINT("\n");
  }

  //set breakpoint here, then analyze results
  return 0;
}

