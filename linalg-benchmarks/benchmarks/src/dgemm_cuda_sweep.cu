#include "hip/hip_runtime.h"
//==========================================================================
// Copyright (c) 2017 Samuel Steffl
// All rights reserved.
//=========================================================================

#include <assert.h>
#include <stdbool.h>
#include <string.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

extern "C" {
#include "get_real_time.h"
#include "matrix_types.h"
#include "matrix_utilities.h"
#include "util.h"
#include "verify/common.h"
}

//==========================================================================

static const double alpha       = 2.5;
static const double beta        = 3.0;

static const IDX NMP_MIN        = 128; //was 2
static const IDX NMP_MAX        = 1024;
static const IDX NMP_SCALE      = 2;

static const bool YES_TRANS     = 1;
static const bool NO_TRANS      = 0;


//A not trans, B not trans
__global__ void MatrixMulKernel_NN(double *dC, double *dA, double *dB,
  IDX N, IDX M, IDX P)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;

  double result = 0;
  for (IDX k=0; k<P; ++k) {
    result += IDXR(dA, i, k, N, P)*IDXR(dB, k, j, P, M);
  }
  IDXR(dC, i, j, N, M) = alpha*result + beta*IDXR(dC, i, j, N, M); 
}

//A not trans, B trans
__global__ void MatrixMulKernel_NT(double *dC, double *dA, double *dB,
  IDX N, IDX M, IDX P)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;

  double result = 0;
  for (IDX k=0; k<P; ++k) {
    result += IDXR(dA, i, k, N, P)*IDXR(dB, j, k, P, M);
  }
  IDXR(dC, i, j, N, M) = alpha*result + beta*IDXR(dC, i, j, N, M); 
}

//A trans, B not trans
__global__ void MatrixMulKernel_TN(double *dC, double *dA, double *dB,
  IDX N, IDX M, IDX P)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;

  double result = 0;
  for (IDX k=0; k<P; ++k) {
    result += IDXR(dA, k, i, N, P)*IDXR(dB, k, j, P, M);
  }
  IDXR(dC, i, j, N, M) = alpha*result + beta*IDXR(dC, i, j, N, M); 
}

//A trans, B trans
__global__ void MatrixMulKernel_TT(double *dC, double *dA, double *dB,
  IDX N, IDX M, IDX P)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;

  double result = 0;
  for (IDX k=0; k<P; ++k) {
    result += IDXR(dA, k, i, N, P)*IDXR(dB, j, k, P, M);
  }
  IDXR(dC, i, j, N, M) = alpha*result + beta*IDXR(dC, i, j, N, M); 
}


//  NO STATIC LINKING IN CUDA 3.2 :(
/*
void use_cublas(double *dC, double *dA, double *dB,
  IDX N, IDX M, IDX P, bool A_trans, bool B_trans)
{
  hipblasDgemm((A_trans ? 'T' : 'N' ),
    (B_trans ? 'T' : 'N'), N, M, P, alpha,
    dA, N, dB, P, beta, dC, N);
}
*/

//According to book, limits are:
//  up to 512 threads per block
//  up to 8 blocks per SM at a time
//  up to 1024 threads per SM
//so: blocks should be 16x16 threads (256 threads) (matrices MUST be 16x16 mul)
//    which satisfies all 3 above
static void do_cuda_dgemm(double *C, double *A, double *B,
  IDX N, IDX M, IDX P, bool A_trans, bool B_trans)
{
  const IDX BLOCK_SIZE = 16;

  double *dC, *dA, *dB;
  IDX C_size = N*M*sizeof(double);
  IDX A_size = N*P*sizeof(double);
  IDX B_size = P*M*sizeof(double);

  //allocate matrices on device
  hipMalloc((void **)&dC, C_size);
  hipMalloc((void **)&dA, A_size);
  hipMalloc((void **)&dB, B_size);

  //transfer A, B, to device
  hipMemcpy(dA, A, A_size, hipMemcpyHostToDevice);
  hipMemcpy(dB, B, B_size, hipMemcpyHostToDevice);
  hipMemcpy(dC, C, C_size, hipMemcpyHostToDevice);

  //perform kernel
  assert(N % BLOCK_SIZE == 0 && M % BLOCK_SIZE == 0);
  dim3 dimGrid(M/BLOCK_SIZE, N/BLOCK_SIZE);
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  if(!A_trans && !B_trans) {
    MatrixMulKernel_NN<<<dimGrid, dimBlock>>>(dC, dA, dB, N, M, P);
  } else if(!A_trans && B_trans) {
    MatrixMulKernel_NT<<<dimGrid, dimBlock>>>(dC, dA, dB, N, M, P);
  } else if(A_trans && !B_trans) {
    MatrixMulKernel_TN<<<dimGrid, dimBlock>>>(dC, dA, dB, N, M, P);
  } else {
    MatrixMulKernel_TT<<<dimGrid, dimBlock>>>(dC, dA, dB, N, M, P);
  }
  //use_cublas(dC, dA, dB, N, M, P, A_trans, B_trans);

  //transfer C to host
  hipMemcpy(C, dC, C_size, hipMemcpyDeviceToHost);  

  //free matrices on device
  hipFree(dC);
  hipFree(dA);
  hipFree(dB);
}


//=============================================================================
//=============================================================================
//=============================================================================
//=============================================================================
//=============================================================================
//=============================================================================
//=============================================================================
//=============================================================================
//=============================================================================
//=============================================================================
//=============================================================================
//=============================================================================
/*
    -- MAGMA (version 0.3) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       June 2010

    -- Developed by:
       Rajib Nath 
       Stan Tomov

    -- The implementation of these DGEMM routines for the Fermi GPUs are described in
       Nath, R., Tomov, S., Dongarra, J. "An Improved MAGMA GEMM for Fermi GPUs," 
       University of Tennessee Computer Science Technical Report, UT-CS-10-655 
       (also LAPACK working note 227), July 29, 2010. 

       These routines will be included in CUBLAS 3.2.
*/

/*
    blk_M=64 blk_N=64 blk_K=16 nthd_x=64 nthd_y=4
*/

//texture<int2,1>  tex_x_double_A;
//texture<int2,1>  tex_x_double_B;

static __inline__ __device__ double fetch_x_A(const int& i, const double *A)
{
  return A[i];
  //register int2  v = tex1Dfetch(tex_x_double_A, i);
  //return __hiloint2double(v.y, v.x);
}
static __inline__ __device__ double fetch_x_B(const int& i, const double *B)
{
  return B[i];
  //register int2  v = tex1Dfetch(tex_x_double_B, i);
  //return __hiloint2double(v.y, v.x);
}


__global__ void fermiDgemm_v2_kernel_NN(double *C, const double *A, const double *B,  
                                        int m, int n, int k, int lda, int ldb,  
                                        int ldc, double alpha, double beta) 
{
    const  int tx = threadIdx.x;
    const  int ty = threadIdx.y;

    const int iby = blockIdx.y * 64;
    const int ibx = blockIdx.x * 64;
    const int idt = ty * 64 + tx;

    const int res = idt%16;
    const int qot = idt/16;

    __shared__ double  Bb[16][65];
    __shared__ double Abs[64][17];

    double xxA[4];
    double xxB[4];
    
    B+= res+ __mul24(iby + qot * 4, ldb );
    int trackB = res+ __mul24(iby + qot * 4, ldb );

    A += ibx +__mul24( qot, lda) + res ; 
    int trackA =  ibx +__mul24( qot, lda) + res ;

    #pragma unroll
    for(int y=0; y<4; y++)
       Bb[res][qot*4+y] = fetch_x_B( trackB + y * ldb, B ) ;

    #pragma unroll
    for(int y=0; y<4; y++)
      Abs[res+ y*16][qot] = fetch_x_A(trackA + y*16, A) ;
    __syncthreads();

    const double *Bend = B + k-16;
   
    double Axs[4];
    double Bxp[4];

    double Cb[16] = {0,0,0,0, 0,0,0,0, 0,0,0,0, 0,0,0,0};

     do 
     {
    B += 16;
    A += lda *16  ;
    trackA += 16*lda ; 
    trackB += 16;

        #pragma unroll
        for( int y=0; y<4; y++)
      xxB[y] = fetch_x_B( trackB + y*ldb, B);

        #pragma unroll
        for( int y=0; y<4; y++)
      xxA[y] = fetch_x_A(trackA + y*16, A);

        #pragma unroll 
        for( int j1=0;j1<16;j1++){

             #pragma unroll
             for( int y=0; y<4; y++)
                  Bxp[y]= Bb[j1][qot+y*16];

             #pragma unroll
             for( int y=0; y<4; y++)
                  Axs[y] =  Abs[res+y*16][j1] ;

             #pragma unroll 
             for( int x=0; x<4; x++){
                  #pragma unroll 
                  for( int y=0; y<4; y++){
                      Cb[x*4+y]  += Axs[x]*Bxp[y];
          }
         }
        }

    __syncthreads();
    #pragma unroll
    for(int y=0; y<4; y++)
        Abs[res+y*16][qot] = xxA[y]; 

    #pragma unroll
    for(int y=0; y<4; y++)
        Bb[res][qot*4 + y] = xxB[y];

    __syncthreads();
     }
     while (B < Bend);

     C += res + ibx  + __mul24 (qot +  iby ,ldc);

     #pragma unroll 
     for(int j1=0;j1<16;j1++){

              #pragma unroll
              for( int y=0; y<4; y++)
                  Bxp[y]= Bb[j1][qot + y*16];

              #pragma unroll
              for( int y=0; y<4; y++)
                  Axs[y] =  Abs[res + y*16][j1] ;

              #pragma unroll 
              for( int x=0; x<4; x++){
                  #pragma unroll 
                  for( int y=0;y<4; y++){
                      Cb[x*4 + y]  += Axs[x]*Bxp[y];
          }
          }
    }

    #pragma unroll
    for( int y=0;y<4;y++){

       #pragma unroll
       for(int x=0;x<4;x++){
          C[x*16] = alpha*Cb[y+x*4] + beta * C[x*16];
       }
       
       C += ldc*16;
    }
}

__global__ void fermiDgemm_v2_kernel_TN(double *C, const double *A, const double *B,
                                        int m, int n,  int k,  int lda,  int ldb,  
                                        int ldc, double alpha, double beta) 
{
    const  int tx = threadIdx.x;
    const  int ty = threadIdx.y;

    const int iby = blockIdx.y * 64;
    const int ibx = blockIdx.x * 64;
    const int idt = ty * 64 + tx;

    const int res = idt%16;
    const int qot = idt/16;

    __shared__ double Bb[16][65];
    __shared__ double Abs[64][17];

    double xxA[4];
    double xxB[4];

    B+= res+ __mul24(iby+qot*4, ldb );
    int trackB = res+ __mul24(iby+qot*4, ldb );

    A+= __mul24( ibx + qot, lda )   + res; 
    int trackA = __mul24( ibx + qot, lda ) + res;

    #pragma unroll
    for(int y=0; y<4; y++)
    Bb[res][qot*4+y] =fetch_x_B( trackB + y*ldb, B );

    #pragma unroll
    for(int y=0; y<4; y++)
    Abs[qot+16*y][res] = fetch_x_A(trackA +  lda*16*y, A);
    __syncthreads();

    const double *Bend = B + k-16;
   
    double Axs[4];
    double Bxp[4];

    double Cb[16] = {0,0,0,0,    0,0,0,0, 0,0,0,0, 0,0,0,0};

    do 
    {
    B += 16;
    A += 16  ;
    trackA+=16 ; 
    trackB+=16;

        #pragma unroll
        for(int y=0; y<4; y++)
       xxB[y]=fetch_x_B( trackB + y*ldb, B);

    #pragma unroll
        for(int y=0; y<4; y++)
       xxA[y] =fetch_x_A(trackA +  lda*y*16, A);

    #pragma unroll 
        for(int j1=0;j1<16;j1++){
                  #pragma unroll
                  for(int y=0; y<4; y++)
                      Bxp[y]= Bb[j1][qot+y*16];

                  #pragma unroll
                  for(int y=0; y<4; y++)
                      Axs[y] =  Abs[res+y*16][j1];

                  #pragma unroll 
                  for(int x=0; x<4; x++){
                     #pragma unroll 
                     for(int y=0; y<4; y++){
                         Cb[x*4+y]  += Axs[x]*Bxp[y];
             }
          }
    }

    __syncthreads();
    #pragma unroll
    for(int y=0; y<4; y++)
       Abs[qot+16*y][res] = xxA[y];
 
    #pragma unroll
    for(int y=0; y<4; y++)
       Bb[res][qot*4+y] =xxB[y];
    __syncthreads();
    } 
    while (B < Bend);

    C += res + ibx  + __mul24 (qot + iby ,ldc);

    #pragma unroll 
    for(int j1=0; j1<16; j1++){
    #pragma unroll
        for(int y=0; y<4; y++)
            Bxp[y]= Bb[j1][qot+y*16];

        #pragma unroll
        for(int y=0; y<4; y++)
            Axs[y] = Abs[res+y*16][j1];

        #pragma unroll 
        for(int x=0; x<4; x++){
            #pragma unroll 
            for(int y=0; y<4; y++){
               Cb[x*4+y] += Axs[x]*Bxp[y];
        }
    }
    }

    #pragma unroll
    for(int y=0;y<4;y++){
    #pragma unroll
        for(int x=0;x<4;x++)
       C[x*16] =alpha*Cb[y+x*4] + beta * C[x*16];
       
    C+=ldc*16;
    }
}


__global__ 
void fermiDgemm_v2_kernel_TT(double *C, const double *A, const double *B,  int m, int n,  
                             int k,  int lda,  int ldb,  int ldc, double alpha, double beta) 
{
    const  int tx = threadIdx.x;
    const  int ty = threadIdx.y;

    const int iby = blockIdx.y * 64;
    const int ibx = blockIdx.x * 64;
    const int idt = ty * 64 + tx;

    const int res = idt% 16;
    const int qot = idt/ 16;

    __shared__ double Bb[16][65];
    __shared__ double Abs[64][17];

    double xxA[4];
    double xxB[4];

    B += iby + tx + __mul24(ty , ldb );
    A += __mul24( ibx + qot , lda )   + res; 

    int trackA = __mul24( ibx + qot, lda ) + res;
    int trackB =  iby+ tx + __mul24(ty , ldb );

    Bb[ty+0*4][tx] = fetch_x_B(trackB+ldb*0, B);
    Bb[ty+1*4][tx] = fetch_x_B(trackB+ldb*4, B);
    Bb[ty+2*4][tx] = fetch_x_B(trackB+ldb*8, B);
    Bb[ty+3*4][tx] = fetch_x_B(trackB+ldb*12, B);
    Abs[qot +16*0][res] = fetch_x_A(trackA +  lda*16*0, A) ;
    Abs[qot +16*1][res] = fetch_x_A(trackA +  lda*16*1, A) ;
    Abs[qot +16*2][res] = fetch_x_A(trackA +  lda*16*2, A) ;
    Abs[qot +16*3][res] = fetch_x_A(trackA +  lda*16*3, A) ;
    __syncthreads();

    const double *Bend = B + k*ldb-16*ldb;
   
    double Axs[4];
    double Bxp[4];

    double Cb[16] = {0,0,0,0,    0,0,0,0, 0,0,0,0, 0,0,0,0};

    do 
    {
    B += 16*ldb;
    A += 16  ;
    trackA+=16 ; 
    trackB+=16*ldb;

    xxB[0]=fetch_x_B( trackB + 0*ldb, B) ;
    xxB[1]=fetch_x_B( trackB + 4*ldb, B) ;
    xxB[2]=fetch_x_B( trackB + 8*ldb, B) ;
    xxB[3]=fetch_x_B( trackB + 12*ldb, B) ;

    xxA[0] =fetch_x_A(trackA +  lda*0*16, A) ;
    xxA[1]=fetch_x_A(trackA +  lda*1*16 , A) ;
    xxA[2]=fetch_x_A(trackA +  lda*2*16 , A) ;
    xxA[3]=fetch_x_A(trackA +  lda*3*16 , A) ;

        #pragma unroll 
        for( int j1=0;j1<16;j1++){
            #pragma unroll
            for( int y=0;y<4;y++)
                Bxp[y]= Bb[j1][qot + y*16];

            #pragma unroll
            for(int y=0;y<4;y++)
                Axs[y] =  Abs[res + y*16][j1];

            #pragma unroll 
            for( int x=0;x<4;x++){
                #pragma unroll 
                for( int y=0;y<4;y++){
                   Cb[x*4+y]  += Axs[x]*Bxp[y];
        }
        }
        }
        
    __syncthreads();
    #pragma unroll
    for(int y=0;y<4;y++)
       Abs[qot+16*y][res] = xxA[y];
 
    #pragma unroll
    for(int y=0;y<4;y++)
       Bb[ty+y*4][tx] =xxB[y];

    __syncthreads();
    } 
    while (B < Bend);

    C += res + ibx  + __mul24 (qot + iby ,ldc);

    #pragma unroll 
    for( int j1=0;j1<16; j1++){
       #pragma unroll
       for( int y=0;y<4;y++)
           Bxp[y]= Bb[j1][qot+y*16];

       #pragma unroll
       for( int y=0;y<4;y++)
           Axs[y] =  Abs[res+y*16][j1];

       #pragma unroll 
       for( int x=0;x<4;x++){
            #pragma unroll 
            for( int y=0;y<4;y++){
               Cb[x*4+y]  += Axs[x]*Bxp[y];
        }
    }
    }

    #pragma unroll
    for( int y=0;y<4;y++){
    #pragma unroll
        for(int x=0; x<4; x++)
       C[x*16] =alpha*Cb[y+x*4] + beta * C[x*16];
           
    C+=ldc*16;
    }
}

__global__ void fermiDgemm_v2_kernel_NT(double *C, const double *A, const double *B,  
                                        int m, int n,  int k,  int lda,  int ldb,  
                                        int ldc, double alpha, double beta) 
{
    const  int tx = threadIdx.x;
    const  int ty = threadIdx.y;

    const int iby = blockIdx.y * 64;
    const int ibx = blockIdx.x * 64;
    const int idt = ty * 64 + tx;

    const int res = idt%16;
    const int qot = idt/16;

    __shared__ double Bb[16][65];
    __shared__ double Abs[64][17];

    double xxA[4];
    double xxB[4];

    B+= iby+ tx + __mul24(ty , ldb );
    int trackB =  iby+ tx + __mul24(ty , ldb );

    A+= ibx +__mul24( qot, lda) + res ; 
    int trackA =  ibx +__mul24( qot, lda) + res ;

    Bb[ty+0*4][tx] = fetch_x_B(trackB+ldb*0, B);
    Bb[ty+1*4][tx] = fetch_x_B(trackB+ldb*4, B);
    Bb[ty+2*4][tx] = fetch_x_B(trackB+ldb*8, B);
    Bb[ty+3*4][tx] = fetch_x_B(trackB+ldb*12, B);

    Abs[res+0*16   ][qot] =fetch_x_A(trackA +  0*16, A) ;
    Abs[res+1*16   ][qot] =fetch_x_A(trackA +  1*16, A) ;
    Abs[res+2*16   ][qot] =fetch_x_A(trackA +  2*16, A) ;
    Abs[res+3*16   ][qot] =fetch_x_A(trackA +  3*16, A) ;
    __syncthreads();

    const double *Bend = B + k*ldb-16*ldb;
   
    double Axs[4];
    double Bxp[4];

    double Cb[16] = {0,0,0,0,    0,0,0,0, 0,0,0,0, 0,0,0,0};

    do 
    {
    B += 16*ldb;
    A += lda *16  ;
    trackA+=16*lda ; 
    trackB+=16*ldb;

    xxB[0]=fetch_x_B( trackB + 0*ldb, B) ;
    xxB[1]=fetch_x_B( trackB + 4*ldb, B) ;
    xxB[2]=fetch_x_B( trackB + 8*ldb, B) ;
    xxB[3]=fetch_x_B( trackB + 12*ldb, B) ;

    xxA[0]=fetch_x_A(trackA +  0*16 , A) ;
    xxA[1]=fetch_x_A(trackA +  1*16 , A) ;
    xxA[2]=fetch_x_A(trackA +  2*16 , A) ;
    xxA[3]=fetch_x_A(trackA +  3*16 , A) ;

        #pragma unroll 
        for( int j1=0;j1<16;j1++){
            #pragma unroll
            for( int y=0;y<4;y++)
                Bxp[y]= Bb[j1][qot+y*16];

            #pragma unroll
            for(int y=0;y<4;y++)
               Axs[y] = Abs[res+y*16][j1] ;

            #pragma unroll 
            for( int x=0;x<4;x++){
                #pragma unroll 
                for( int y=0;y<4;y++){
                   Cb[x*4+y]  += Axs[x]*Bxp[y];
        }
        } 
        }
        
    __syncthreads();
    #pragma unroll
    for(  int y=0;y<4;y++)
        Abs[res+y*16   ][qot] =xxA[y]; 

    #pragma unroll
    for(  int y=0;y<4;y++)
        Bb[ty+y*4][tx] =xxB[y];
    __syncthreads();
    } 
    while (B < Bend);

    C += res + ibx  + __mul24 (qot + iby ,ldc);

    #pragma unroll 
    for( int j1=0;j1<16;j1++){
        #pragma unroll
        for( int y=0;y<4;y++)
            Bxp[y]= Bb[j1][qot+y*16];

        #pragma unroll
        for( int y=0;y<4;y++)
            Axs[y] =  Abs[res+y*16][j1];

        #pragma unroll 
        for( int x=0; x<4; x++){
            #pragma unroll 
            for( int y=0;y<4;y++){
                 Cb[x*4+y]  += Axs[x]*Bxp[y];
        }
    }
    }

    #pragma unroll
    for( int y=0;y<4;y++){
    #pragma unroll
        for(int x=0;x<4;x++)
       C[x*16] =alpha*Cb[y+x*4] + beta * C[x*16];
       
    C+=ldc*16;
    }
}

extern "C" void
magmablas_fermi_dgemm(char TRANSA, char TRANSB, int m , int n , int k , 
                      double alpha, const double *A, int lda, const double *B, 
                      int ldb, double beta, double *C, int ldc ) 
{
   if (m<=0 || n<=0 || k<=0)
     return;

   /*
   if( m % (64) !=0 || n% (64)!=0 || k%(64) !=0 )
   {
    printf("Dimension Should Be multiple of %d\n", 64);
    printf("Calling hipblasDgemm\n");
    hipblasDgemm(TRANSA, TRANSB, m, n, k, alpha, A, lda, B,ldb,
                    beta, C, ldc);
    return;
   }
   */
   assert(m%64 == 0 && n%64==0 && k%64==0);

   //hipChannelFormatDesc channelDesc = 
   //    hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindSigned);
   //hipError_t  errt = hipBindTexture(0,tex_x_double_A,A,channelDesc);
   //if( errt != hipSuccess) printf("can not bind to texture \n");
   //errt = hipBindTexture(0,tex_x_double_B,B,channelDesc);
   //if( errt != hipSuccess) printf("can not bind to texture \n");

   dim3 threads( 64, 4 );
   dim3 grid(m/(64)+(m%(64)!=0),n/(64)+(n%(64)!=0));

  if( TRANSB == 'T' || TRANSB == 't') 
    if( TRANSA == 'N' ||  TRANSA == 'n') 
      fermiDgemm_v2_kernel_NT<<< grid, threads>>>(C, A, B, m, n, k, lda, ldb, 
                                                  ldc, alpha, beta);
    else
      fermiDgemm_v2_kernel_TT<<< grid, threads>>>(C, A, B, m, n, k, lda, ldb, 
                                                  ldc, alpha, beta);
  else
    if( TRANSA == 'N' || TRANSA == 'n') 
      fermiDgemm_v2_kernel_NN<<< grid, threads>>>(C, A, B, m, n, k, lda, ldb, 
                                                  ldc, alpha, beta);
    else
      fermiDgemm_v2_kernel_TN<<< grid, threads>>>(C, A, B, m, n, k, lda, ldb, 
                                                  ldc, alpha, beta);

   //hipUnbindTexture ( tex_x_double_A ) ;
   //hipUnbindTexture ( tex_x_double_B ) ;
}

//=============================================================================
//=============================================================================
//=============================================================================
//=============================================================================
//=============================================================================
//=============================================================================
//=============================================================================
//=============================================================================
//=============================================================================
//=============================================================================
//=============================================================================
//=============================================================================

//all must be column matrices!
static void do_magma_dgemm(double *C, double *A, double *B,
  IDX N, IDX M, IDX P, bool A_trans, bool B_trans)
{
  double *dC, *dA, *dB;
  IDX C_size = N*M*sizeof(double);
  IDX A_size = N*P*sizeof(double);
  IDX B_size = P*M*sizeof(double);

  //allocate matrices on device
  hipMalloc((void **)&dC, C_size);
  hipMalloc((void **)&dA, A_size);
  hipMalloc((void **)&dB, B_size);

  //transfer A, B, to device
  hipMemcpy(dA, A, A_size, hipMemcpyHostToDevice);
  hipMemcpy(dB, B, B_size, hipMemcpyHostToDevice);
  hipMemcpy(dC, C, C_size, hipMemcpyHostToDevice);

  magmablas_fermi_dgemm((A_trans ? 'T' : 'N'), (B_trans ? 'T' : 'N'), N, M, P, 
    alpha, dA, N, dB, P, beta, dC, N);

  //transfer C to host
  hipMemcpy(C, dC, C_size, hipMemcpyDeviceToHost);  

  //free matrices on device
  hipFree(dC);
  hipFree(dA);
  hipFree(dB);
}

static void do_test(IDX N, IDX M, IDX P, bool A_trans, bool B_trans)
{
  double *C, *A, *B;
  C = create_matrix_dc(N, M, FILL);
  A = create_matrix_dc(N, P, FILL);
  B = create_matrix_dc(P, M, FILL);

  double flop = GEMM_FLOP(N, M, P);

  double magma_start = get_real_time();
  do_magma_dgemm(C, A, B, N, M, P, A_trans, B_trans);
  double magma_duration = get_real_time() - magma_start;

  PRINT("N=%4d M=%4d P=%4d (%s:%s) | MAGMA took % 10.6f secs | ", 
    N, M, P,
    (A_trans ? "Atrns" : "Anorm"), (B_trans ? "Btrns" : "Bnorm"), magma_duration);
  PRINT("% 9.3f MFLOP | ", flop / 1e6);
  PRINT("% 12.3f MFLOP/s\n", flop/ magma_duration / 1e6);

  double naive_start = get_real_time();
  do_cuda_dgemm(C, A, B, N, M, P, A_trans, B_trans);
  double naive_duration = get_real_time() - naive_start;

  PRINT("N=%4d M=%4d P=%4d (%s:%s) | NAIVE took % 10.6f secs | ", 
    N, M, P,
    (A_trans ? "Atrns" : "Anorm"), (B_trans ? "Btrns" : "Bnorm"), naive_duration);
  PRINT("% 9.3f MFLOP | ", flop / 1e6);
  PRINT("% 12.3f MFLOP/s\n", flop/ naive_duration / 1e6);

  printf("results match\n");

  free_matrix_d(C);
  free_matrix_d(A);
  free_matrix_d(B);
}

int main(int argc, char **argv)
{
  //---------------------------------------------------------------------------
  unsigned seed = time(NULL);
  IDX nmp_start = NMP_MIN;
  IDX nmp_end = NMP_MAX;
  IDX idx = -1;

  unsigned tmp_seed;
  IDX tmp_nmp;
  IDX tmp_idx;

  //---------------------------------------------------------------------------
  //parse args
  for(int i=1; i<argc; ++i){
    if(sscanf(argv[i], "--seed=%u", &tmp_seed) == 1){
      seed = tmp_seed;
    }
    else if(sscanf(argv[i], "--size=%u", &tmp_nmp) == 1){
      nmp_start = tmp_nmp;
      nmp_end = tmp_nmp;
    }
    else if(sscanf(argv[i], "--idx=%d", &tmp_idx) == 1){
      if(tmp_idx > 3 || tmp_idx < 0) {
        printf("invalid idx, must be [0,3]\n");
        exit(0);
      }
      idx = tmp_idx;
    }
    else {
      printf("unrecognized argument: %s\n", argv[i]);
      exit(0);
    }
  }

  //---------------------------------------------------------------------------
  //just to kinda randomize the matrices up
  srand(seed);

  //---------------------------------------------------------------------------

  //set breakpoint here, then make command window huge

  for(IDX NMP=nmp_start; NMP <= nmp_end; NMP *= NMP_SCALE){
    if(idx == -1) {
      do_test(NMP, NMP, NMP, NO_TRANS,  NO_TRANS);
      do_test(NMP, NMP, NMP, NO_TRANS,  YES_TRANS);
      do_test(NMP, NMP, NMP, YES_TRANS, NO_TRANS);
      do_test(NMP, NMP, NMP, YES_TRANS, YES_TRANS);
    } else if(idx == 0) {
      do_test(NMP, NMP, NMP, NO_TRANS,  NO_TRANS);
    } else if(idx == 1) {
      do_test(NMP, NMP, NMP, NO_TRANS,  YES_TRANS);
    } else if(idx == 2) {
      do_test(NMP, NMP, NMP, YES_TRANS, NO_TRANS);
    } else {
      do_test(NMP, NMP, NMP, YES_TRANS, YES_TRANS);
    }

    PRINT("\n");
  }

  //set breakpoint here, then analyze results
  return 0;
}
